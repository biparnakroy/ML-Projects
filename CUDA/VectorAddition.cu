
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE	1024

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i= threadIdx.x;

	if(i<n)
		c[i] = a[i] + b[i];
}

int main()
{
	int *a, *b, *c;
	
	hipMallocManaged(&a, SIZE * sizeof(int));
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));
	
	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}
	
    VectorAdd <<<1, SIZE>>> (a, b, c, SIZE);
    
    hipDeviceSynchronize();

	for (int i = 0; i < 10; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}